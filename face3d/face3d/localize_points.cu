#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "triangle_mesh.h"

__global__ void localize_points(float* V, float* d_query_points, float* d_query_points_xy,
                                float* d_locs, float* d_dists, int nvertices, int nquery_points,
                                int query_points_size, int query_points_xy_size, int threads_per_block) {

  int index = blockDim.x * blockIdx.x + threadIdx.x;
  float min_dist = 100;
  float minx, miny;
  float x, y, z, r, g, b, dist;
  int start, end;
  int shared_arr_size = 1024;
  __shared__ float l_query_points[1024*3];

  x = V[3*index];
  y = V[3*index+1];
  z = V[3*index+2];

  for (int i = 0; i < nquery_points; i += shared_arr_size) {
    start = ceilf(shared_arr_size/(threads_per_block*1.0))*threadIdx.x;
    end = min(ceilf(shared_arr_size/(threads_per_block*1.0))*(threadIdx.x+1), float(shared_arr_size));

    for (int j = start; j < end && i+j < nquery_points; j++) {
      l_query_points[j*3] = d_query_points[(i+j)*3];
      l_query_points[j*3+1] = d_query_points[(i+j)*3+1];
      l_query_points[j*3+2] = d_query_points[(i+j)*3+2];
    }
    __syncthreads();

    for (int j = 0; j < shared_arr_size && i+j < nquery_points; j++) {
      r = l_query_points[j*3];
      g = l_query_points[j*3+1];
      b = l_query_points[j*3+2];
      dist = (x-r)*(x-r) + (y-g)*(y-g) + (z-b)*(z-b);
      if (dist < min_dist) {
        min_dist = dist;
        minx = d_query_points_xy[(i+j)*2];
        miny = d_query_points_xy[(i+j)*2+1];
      }
    }
    __syncthreads();
  }

  if (index < nvertices) {
    d_locs[2*index] = minx;
    d_locs[2*index+1] = miny;
    d_dists[index] = min_dist;
  }
  return;
}

namespace face3d {
  void localize_points_cuda(MatrixXfRowMajor& query_points,
                            MatrixXfRowMajor& query_points_xy,
                            MatrixXfRowMajor& V,
                            MatrixXfRowMajor& locs_eigen,
                            MatrixXfRowMajor& dists,
                            int cuda_device
                            ) {

    int V_size = V.rows()*V.cols()*sizeof(float);
    int query_points_size = query_points.rows()*query_points.cols()*sizeof(float);
    int query_points_xy_size = query_points_xy.rows()*query_points_xy.cols()*sizeof(float);
    int locs_size = locs_eigen.rows()*locs_eigen.cols()*sizeof(float);
    int dists_size = locs_eigen.rows()*sizeof(float);

    if(hipSetDevice(cuda_device) != hipSuccess) {
      std::cerr << "ERROR setting cuda device to " << cuda_device << std::endl;
      throw std::runtime_error("hipSetDevice returned error");
    }

    float *d_V, *d_query_points, *d_query_points_xy, *d_locs, *d_dists;

    hipMalloc((void **)&d_V, V_size);
    hipMalloc((void **)&d_query_points, query_points_size);
    hipMalloc((void **)&d_query_points_xy, query_points_xy_size);
    hipMalloc((void **)&d_locs, locs_size);
    hipMalloc((void **)&d_dists, dists_size);

    hipMemcpy(d_V, V.data(), V_size, hipMemcpyHostToDevice);
    hipMemcpy(d_query_points, query_points.data(), query_points_size, hipMemcpyHostToDevice);
    hipMemcpy(d_query_points_xy, query_points_xy.data(), query_points_xy_size, hipMemcpyHostToDevice);

    hipMemset(d_locs, 0, locs_size);
    hipMemset(d_dists, 0, dists_size);

    int nvertices = V.rows();
    int nquery_points = query_points.rows();

    int threads_per_block = 256;
    int blocks_per_grid = (nvertices + threads_per_block - 1) / threads_per_block;

    localize_points<<<blocks_per_grid, threads_per_block>>> (d_V, d_query_points, d_query_points_xy,
                                                             d_locs, d_dists, nvertices, nquery_points,
                                                             query_points_size, query_points_xy_size, threads_per_block);

    hipDeviceSynchronize();

    hipMemcpy(locs_eigen.data(), d_locs, locs_size, hipMemcpyDeviceToHost);
    hipMemcpy(dists.data(), d_dists, dists_size, hipMemcpyDeviceToHost);

    hipFree(d_V); hipFree(d_locs);
    hipFree(d_query_points); hipFree(d_query_points_xy);
    hipFree(d_dists);
  }
}
